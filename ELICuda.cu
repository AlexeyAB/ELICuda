#include "hip/hip_runtime.h"
// ELICuda.cpp: ���������� ����� ����� ��� ����������� ����������.
//

// user settings
bool wait_in = true;			// to expect before the exit
const size_t count_it = 5000;	// count of iterations of the loop (division) Repeated division leads to zero results!!!
//#undef __HIPCC__				// or CUDA_ARCH when uses nvcc - compiler. Use CUDA for OpenMP/GPGPU
#define USE_THRUST				// for use Thrust OMP and may be CUDA

#ifdef __HIPCC__
	#define USE_THRUST
	const bool use_cuda = true;
#else
	const bool use_cuda = false;
#endif
//---------------------------------------------------------------------------
		

// std::
#include <iostream>
#include <string>
#include <ctime>
#include <climits>


// thrust::
#ifdef USE_THRUST 
	// For Visual Studio 2010 and CUDA 4.2:
	// In Project Properties -> Configuration Properties -> CUDA C/C++ -> Command Line -> Additional Options: -Xcompiler "/openmp"
	#ifdef _OPENMP
		#define THRUST_HOST_SYSTEM THRUST_HOST_SYSTEM_OMP	// nvcc -Xcompiler "/openmp"
		const bool use_omp = true;
	#else
		const bool use_omp = false;
	#endif
	//#define THRUST_HOST_SYSTEM THRUST_HOST_SYSTEM_TBB

	#include "hip/hip_runtime.h"
	#include ""
	#include <thrust/version.h>
	#include <thrust/host_vector.h>
	#include <thrust/device_vector.h>
	#include <thrust/copy.h>
	#include <thrust/transform.h>
	#include <thrust/functional.h>
	
	namespace eli {
		using namespace thrust;
		typedef host_vector<unsigned short int> T_use_h_vec;
#ifdef __HIPCC__
		typedef device_vector<unsigned short int> T_use_d_vec;
#else
		typedef host_vector<unsigned short int> T_use_d_vec;
#endif
		static_assert(sizeof(unsigned short int) == 2, "Must 1 pixel = 2 bytes!");
	};
#else
// std::
	const bool use_omp = false;
	//#include <omp.h>
	#include <vector>
	#include <algorithm>
	#include <functional>
	#define THRUST_MAJOR_VERSION 0
	#define THRUST_MINOR_VERSION 0

	namespace eli {
		using namespace std;
		typedef vector<unsigned short int> T_use_h_vec;
		typedef vector<unsigned short int> T_use_d_vec;
		static_assert(sizeof(unsigned short int) == 2, "Must 1 pixel = 2 bytes!");
	};
#endif


#include "ELIdivision.h"
//---------------------------------------------------------------------------

// task of division images
template<typename T>
bool division(const std::string input_file1, const std::string input_file2, const std::string output_file, const size_t _count_it) {

	try {

		eli::T_ELI_file<T> eli_file1(input_file1);
		std::cout << "Loaded ELI file: " << input_file1 << std::endl;
		eli::T_ELI_file<T> eli_file2(input_file2);
		std::cout << "Loaded ELI file: " << input_file2 << std::endl;

		eli_file1.division(eli_file2);
		std::cout << "Divided" << std::endl;
		if (eli_file1.save_eli_file(output_file)) std::cout << "ELI saved to file: " << output_file << std::endl;
		else std::cout << "Failed save EIL to file: " << output_file << std::endl;

		std::cout << "Process of dividing is performed for the number of iterations equal to: " << _count_it << std::endl;
		std::cout << "in progress..." << std::endl;
		clock_t t1, t2;
		t1 = clock();

		for(int i = 0; i < _count_it; ++i)
			eli_file1.division(eli_file2);

		t2 = clock();
		std::cout << "Time: " << static_cast<float>(t2 - t1)/CLOCKS_PER_SEC << " seconds" << std::endl;



	} catch(...) {
		std::cout << eli::exception_catch("Error catch at: " + THROW_PLACE) << std::endl << std::endl;
		return false;
	}
	return true;
}
//---------------------------------------------------------------------------


int main(int argc, char * argv[])
{
    if(THRUST_MAJOR_VERSION) std::cout << "Thrust v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << std::endl;

	std::string input_file1, input_file2, output_file;
	//input_file2 = input_file1 = "LENA16.ELI";

	if (argc == 3) {
		input_file1 = argv[1];
		input_file2 = argv[2];

		if(use_omp) std::cout << std::endl << "Use OpenMP";
		std::cout << std::endl << "On CPU:" << std::endl;
		if(division<eli::T_use_h_vec>(input_file1, input_file2, "out_cpu.eli", count_it) ) std::cout << "Done!" << std::endl;
		else std::cout << "Failed!" << std::endl;

		if (use_cuda) {
			std::cout << std::endl << "On GPU:" << std::endl;
			if(division<eli::T_use_d_vec>(input_file1, input_file2, "out_gpu.eli", count_it) ) std::cout << "Done!" << std::endl;
			else std::cout << "Failed!" << std::endl;
		}
	} else {
		std::cout << std::endl << "Arguments not found!" << std::endl << "Use an example: ELICuda.exe filename1 filename2" << std::endl;
	}

	if (wait_in) std::cin >> wait_in;
	return 0;
}

